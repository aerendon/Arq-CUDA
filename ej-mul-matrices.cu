//http://cocinando-tarjetas-graficas.blogspot.com.co/2013/09/multiplicar-matrices-en-cuda.html

#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 16

void matrixMultCPU(int a[N][N], int b[N][N], int c[N][N]) {
int n,m;
 for (int i = 0; i < N; i++) {
  for (int j = 0; j < N; j++) {
   int sum = 0;
   for (int k = 0; k < N; k++) {
    m = a[i][k];
    n = b[k][j];
    sum += m * n;
   }
   c[i][j] = sum;
  }
 }
}

__global__ void matrixMultGPU(int *a, int *b, int *c) {
 int k, sum = 0;
 int col = threadIdx.x + blockDim.x * blockIdx.x;
 int fil = threadIdx.y + blockDim.y * blockIdx.y;

 if (col < N && fil < N) {
  for (k = 0; k < N; k++) {
   sum += a[fil * N + k] * b[k * N + col];
  }
  c[fil * N + col] = sum;
 }
}

int main() {
 int a[N][N], b[N][N], c[N][N];
 int *dev_a, *dev_b, *dev_c;
 int cont,i,j;

 /* inicializando variables con datos foo*/
 for (i = 0; i < N; i++) {
  cont = 0;
  for (j = 0; j < N; j++) {
   a[i][j] = cont;
   b[i][j] = cont;
   cont++;
  }
 }

 int size = N * N * sizeof(int);

 hipMalloc((void **) &dev_a, size);
 hipMalloc((void **) &dev_b, size);
 hipMalloc((void **) &dev_c, size);

 hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
 hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

 dim3 dimGrid(1, 1);
 dim3 dimBlock(N, N);

 matrixMultGPU<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c);

 hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

 hipFree(dev_a);
 hipFree(dev_b);
 hipFree(dev_c);

 // imprimiendo
 for (int y = 0; y < N; y++) {
  for (int x = 0; x < N; x++) {
   printf("[%d][%d]=%d ", y, x, c[y][x]);
  }
  printf("\n");
 }

 return 0;

}