#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#define N 7

using namespace std;

void multMatCPU(int *A, int *B,int *C){
	int aux = 0;
	for(int i=0; i<N; i++){
	    for(int j=0; j<N; j++){
		    aux = 0;
		    for(int k=0; k<N; k++)
		        aux += A[i*N+k]* B[k*N+j];
		    C[i*N+j] = aux;
	    }
	}
}

__global__ void multMatGPU(int *d_A, int *d_B,int *d_C){
	int i = blockIdx.y*blockDim.y+threadIdx.y;
	int j = blockIdx.x*blockDim.x+threadIdx.x;
	if(i < N && j < N){
	    int Pvalue = 0;
	    for(int k=0; k<N; k++){
	       	Pvalue += d_A[i*N+k] * d_B[k*N+j];
	    }
	    d_C[i*N+j] = Pvalue;
	}
}

/**
 * [fill: Llena un array bidimensional con el número 5]
 * @param Matrix [Array[][]]
 */
void fill(int* Matrix){
	for(int i=0; i<N; i++){
	    for(int j=0; j<N; j++){
	      	Matrix[i*N+j] = 5;
	    }
	}
}

/**
 * [display: Muestra un array bidimensional]
 * @param Matrix [Array[][]]
 */
void display(int *Matrix){
	for(int i=0; i<N; i++){
	    for(int j=0; j<N; j++){
	      	cout<<Matrix[i*N+j]<<" ";
	    }
	    cout<<endl;
	}
}


int main(){ 
	int *x = (int*)malloc(N*N*sizeof(int));
	int *y = (int*)malloc(N*N*sizeof(int));
	int *z = (int*)malloc(N*N*sizeof(int));
	//Reloj
	clock_t start, end;
	double cpu_time, gpu_time;

	//Matrices Iniciales
	fill(x);
	fill(y);

	//-------------CPU------------------------
	start = clock();
	multMatCPU(x,y,z);
	end = clock();

	//Mostrar Matrices
	//display(x); display(y); display(z);

	cpu_time = ((double) (end - start)) / CLOCKS_PER_SEC;
	printf("CPU = %lf s\n",cpu_time);

	//-------------GPU------------------------
	int *d_x,*d_y,*d_z;
	float blockSize = 32;
	dim3 dimBlock(blockSize,blockSize);
	dim3 dimGrid(ceil(N/float(blockSize)),ceil(N/float(blockSize)),1);

	hipMalloc((void**)&d_x,sizeof(int)*N*N);
	hipMalloc((void**)&d_y,sizeof(int)*N*N);
	hipMalloc((void**)&d_z,sizeof(int)*N*N);

	start = clock();

	hipMemcpy(d_x, x, sizeof(int)*N*N,hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, sizeof(int)*N*N,hipMemcpyHostToDevice);


	multMatGPU<<<dimGrid,dimBlock>>>(d_x, d_y, d_z);
	hipMemcpy(z,d_z,sizeof(int)*N*N,hipMemcpyDeviceToHost);
	end = clock();
	gpu_time = ((double) (end - start)) / CLOCKS_PER_SEC;
	printf("GPU = %lf s\n",gpu_time);

	//display(z);
	
	//Liberar Memoria
	free(x);
	free(y);
	free(z);

	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_z);

	return 0;  
}