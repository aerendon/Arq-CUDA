#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
//Height and Weight (Tamaño de la Matriz)
#define N 10

using namespace std;

void fill(int *Matrix){
    for(int i=0; i<N; i++){
        for(int j=0; j<N; j++){
            Matrix[(i*N)+j] = j;
        }
    }
}

void display(int *Matrix){
    for(int i=0; i<N; i++){
        for(int j=0; j<N; j++){
            cout<<Matrix[i*N+j]<<" ";
        }
        cout<<endl;
    }
    cout<<endl;
}

//Multiplicación CPU
void mult(int *A, int *B,int *C){
	int aux = 0;
	for(int i=0; i<N; i++){
		for(int j=0; j<N; j++){
			aux = 0;
			for(int k=0; k<N; k++)
			    aux += A[i*N+k]* B[k*N+j];
			C[i*N+j] = aux;
		}
	}
}

//Multiplicación GPU
__global__ void multMat(int *a, int *b, int *c){
	int k, value = 0;
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	if(col < N && row < N) {
		for(k = 0; k < N; k++) value += a[row * N +k] * b[k * N + col];
		c[row * N +col]	 = value;	
	}
}

int main(){
	int *A = (int*)malloc(N*N*sizeof(int));
	int *B = (int*)malloc(N*N*sizeof(int));
	int *C = (int*)malloc(N*N*sizeof(int));
	int *D = (int*)malloc(N*N*sizeof(int));

	//Creacion de las matrices
	fill(A);
	fill(B);

	mult(A, B, C); //CPU

	//Muestra el contenido de las matrices
	display(A);
	display(B);
	display(C);

	return 0;
}